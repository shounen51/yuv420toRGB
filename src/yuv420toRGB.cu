#include "hip/hip_runtime.h"
#include "YUV420ToRGB.h"
#include "cuda_utils.h"
#include ""
#include <iostream>

static uint8_t* img_buffer_host = nullptr;
static uint8_t* img_buffer_device = nullptr;

__global__ void yuv420toRGBKernel(uint8_t* yuv420, int width, int height, uint8_t* rgb) {
    // sperate Y, U, and V planes
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int pixels = width * height;
    if (i >= pixels) return;
    uint8_t* y = yuv420 + i;
    uint8_t* u = yuv420 + width * height + (i / width / 2) * (width / 2)  + i % width / 2;
    uint8_t* v = yuv420 + width * height + (width * height / 4) + (i / width / 2) * (width / 2) + i % width / 2;
    int r = *y + 1.402 * (*v - 128);
    int g = *y - 0.344136 * (*u - 128) - 0.714136 * (*v - 128);
    int b = *y + 1.772 * (*u - 128);
    rgb[i * 3] = min(max(r, 0), 255);
    rgb[i * 3 + 1] = min(max(g, 0), 255);
    rgb[i * 3 + 2] = min(max(b, 0), 255);
}

void yuv420toRGBInPlace(uint8_t* yuv420, int width, int height, uint8_t* rgb, hipStream_t stream) {
    int img_size = width * height * 3 / 2;

    if (img_buffer_host == nullptr) {
        img_buffer_host = new uint8_t[img_size];
    }
    if (img_buffer_device == nullptr) {
        hipMalloc(&img_buffer_device, img_size);
    }

    memcpy(img_buffer_host, yuv420, img_size);

    // Copy YUV420 data to device
    hipMemcpy(img_buffer_device, img_buffer_host, img_size, hipMemcpyHostToDevice);

    // Launch kernel
    int pixels = width * height;
    int numThreads = 256;
    int numBlocks = ceil(pixels / (float)numThreads);
    std::cout << "yuv420toRGBKernel start." << std::endl;
    yuv420toRGBKernel<<<numBlocks, numThreads, 0 , stream>>>(img_buffer_device, width, height, rgb);
    std::cout << "yuv420toRGBKernel done." << std::endl;
    // Add error checking after kernel launch
    hipError_t err = hipGetLastError();
    if (err != hipSuccess) {
        std::cerr << "CUDA kernel launch error: " << hipGetErrorString(err) << std::endl;
    }
    err = hipStreamSynchronize(stream);
    if (err != hipSuccess) {
        std::cerr << "CUDA stream sync error: " << hipGetErrorString(err) << std::endl;
    }
}